#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <glib.h>
//#include<cuda_util.h>

#define R 0
#define G 1
#define B 2
#define RGB 3

typedef guint8 pixel;
#define P 16
#define pixel4 char4

#define  odd(n) ((n)&1)
#define even(n) (!odd((n)))

#define get_filter_color(x,y) (even(x) ? (even(y) ? R : G) : (odd(y) ? B : G))

//texture<pixel4, 2, hipReadModeElementType> src_g;
texture<float4, 2, hipReadModeElementType> horz_tex;
texture<float4, 2, hipReadModeElementType> vert_tex;
texture<guint8, 2, hipReadModeElementType> homo_h_tex;
texture<guint8, 2, hipReadModeElementType> homo_v_tex;

//#define tex_get_comp(tex,x,y,c) tex2D((src),(mirror((x),width))*3+(c),(mirror((y),height)))
//#define tex_get_comp(tex,x,y,c) tex2D((tex),(x)*3+(c),(y))
#define tex_get_color(tex,x,y,c) tex2D((tex),((x)*3)+(c),(y))
#define texR(tex,x,y) (tex2D<pixel4>((tex),(x),(y)).x)
#define texG(tex,x,y) (tex2D<pixel4>((tex),(x),(y)).y)
#define texB(tex,x,y) (tex2D<pixel4>((tex),(x),(y)).z)

#define cR(c4) (c4.x)
#define cG(c4) (c4.y)
#define cB(c4) (c4.z)

#define clampc(a) ((a) < 0) ? 0 : (((a) > 255) ? 255 : (guchar)(a))
#define tex2Du8 tex2D<guint8>

__global__ void ahd_kernel_interp_g(hipTextureObject_t src, pixel4* g_horz_res, pixel4* g_vert_res, int width, int height)
{
  uint x = blockIdx.x*blockDim.x + threadIdx.x;
  uint y = blockIdx.y*blockDim.y + threadIdx.y;
  if (x < 2 || y < 2 || x >= width-2 || y >= height-2) {
    return;
  }
  int filter_color = get_filter_color(x,y);

  char4 h_res, v_res;
  /* Copy existing value to output */
  cR(h_res) = cR(v_res) = (filter_color == R) * tex2Du8(src,float(x),float(y));
  cG(h_res) = cG(v_res) = (filter_color == G) * tex2Du8(src,float(x),float(y));
  cB(h_res) = cB(v_res) = (filter_color == B) * tex2Du8(src,float(x),float(y));


  /* Interpolate Green values first */
  if (filter_color == R || filter_color == B) {
    /* Filter color is red or blue Interpolate green channel horizontally */
    /* Use existing green values */
    float sum = (tex2Du8(src,x-1,y) +
                 tex2Du8(src,x+1,y))/2.0f;

    /* And use existing red/blue values and apply filter 'h' */
    sum += (-tex2Du8(src,x-2,y)/4.0f +
             tex2Du8(src,x,  y)/2.0f +
            -tex2Du8(src,x+2,y)/4.0f)/4.0f;

    cG(h_res) = (guchar)clampc(sum);

    /* Interpolate green channel vertically */
    /* Use existing green values */
    sum = (tex2Du8(src,x,y-1) +
           tex2Du8(src,x,y+1))/2.0f;

    /* And use existing red/blue values and apply filter 'h' */
    sum += (-tex2Du8(src,x,y-2)/4.0f +
             tex2Du8(src,x,y  )/2.0f +
            -tex2Du8(src,x,y+2)/4.0f)/4.0f;

    cG(v_res) = (guchar)clampc(sum);
  }
  int res_index = (y*width + x);
  g_horz_res[res_index] = h_res;
  g_vert_res[res_index] = v_res;
}

#define  labXr_32f  0.433953f /* = xyzXr_32f / 0.950456 */
#define  labXg_32f  0.376219f /* = xyzXg_32f / 0.950456 */
#define  labXb_32f  0.189828f /* = xyzXb_32f / 0.950456 */

#define  labYr_32f  0.212671f /* = xyzYr_32f */
#define  labYg_32f  0.715160f /* = xyzYg_32f */
#define  labYb_32f  0.072169f /* = xyzYb_32f */

#define  labZr_32f  0.017758f /* = xyzZr_32f / 1.088754 */
#define  labZg_32f  0.109477f /* = xyzZg_32f / 1.088754 */
#define  labZb_32f  0.872766f /* = xyzZb_32f / 1.088754 */

#define  labRx_32f  3.0799327f  /* = xyzRx_32f * 0.950456 */
#define  labRy_32f  (-1.53715f) /* = xyzRy_32f */
#define  labRz_32f  (-0.542782f)/* = xyzRz_32f * 1.088754 */

#define  labGx_32f  (-0.921235f)/* = xyzGx_32f * 0.950456 */
#define  labGy_32f  1.875991f   /* = xyzGy_32f */
#define  labGz_32f  0.04524426f /* = xyzGz_32f * 1.088754 */

#define  labBx_32f  0.0528909755f /* = xyzBx_32f * 0.950456 */
#define  labBy_32f  (-0.204043f)  /* = xyzBy_32f */
#define  labBz_32f  1.15115158f   /* = xyzBz_32f * 1.088754 */

#define  labT_32f   0.008856f

#define cvCbrt(value) (__powf(value,1.0f/3.0f))

#define labSmallScale_32f  7.787f
#define labSmallShift_32f  0.13793103448275862f  /* 16/116 */
#define labLScale_32f      116.f
#define labLShift_32f      16.f
#define labLScale2_32f     903.3f

__global__  void ahd_kernel_interp_rb(hipTextureObject_t src_g,float4* g_result, pixel *g_tmp_result, int width, int height) {
  uint x = blockIdx.x*blockDim.x + threadIdx.x;
  uint y = blockIdx.y*blockDim.y + threadIdx.y;

  // Take account of padding in source image
  x += P;
  y += P;

  if (x >= width-P || y >= height-P) {
    return;
  }
  pixel pixR = texR(src_g,x,y);
  pixel pixB = texB(src_g,x,y);

  guchar filter_color = get_filter_color(x,y);

  if (filter_color == R || filter_color == B) {
    /* Filter color is red or blue, interpolate missing red or blue channel */
    /* This function operates the same for horiz and vert interpolation */

    int dest_color = (filter_color == R) ? B : R;
    /* Get the difference between the Red/Blue and Green
     * channels */
    float sum =   (-texG(src_g,x-1,y-1)) +
                  (-texG(src_g,x-1,y+1)) +
                  (-texG(src_g,x+1,y-1)) +
                  (-texG(src_g,x+1,y+1));
  if (dest_color == R) {
    sum += texR(src_g,x-1,y-1) +
           texR(src_g,x-1,y+1) +
           texR(src_g,x+1,y-1) +
           texR(src_g,x+1,y+1);
  } else {
    sum += texB(src_g,x-1,y-1) +
           texB(src_g,x-1,y+1) +
           texB(src_g,x+1,y-1) +
           texB(src_g,x+1,y+1);
    }
    /* Apply low pass filter to the difference */
    sum /= 4.0;
    /* Use interpolated or interpolated green value */
    sum += texG(src_g,x,y);
    pixel res = clampc(round(sum));
    if (filter_color == R) {
      pixR = texR(src_g,x,y);
      pixB = res;
    } else {
      pixB = texB(src_g,x,y);
      pixR = res;
    }
    //res_pix[dest_color] = clampc(round(sum));
  } else {
    /* Filter color is green */
    /* Interpolate Red and Blue channels */
    /* This function operates the same for horz and vert interpolation */
    float sum = 0;
    /* Interpolate Red */
    if (even(y)){
      /* Red/Green rows */
      /* Use left and right pixels */
      /* Get the difference between the Red and Green
       * channel (use only the sampled Green values) */
      sum = (texR(src_g,x-1,y) - texG(src_g,x-1,y)) +
            (texR(src_g,x+1,y) - texG(src_g,x+1,y));
    } else {
      /* Blue/Green rows */
      /* Use top and bottom values */
      sum = (texR(src_g,x,y-1) - texG(src_g,x,y-1)) +
            (texR(src_g,x,y+1) - texG(src_g,x,y+1));
    }
    /* Apply low pass filter */
    sum /= 2.0;
    sum += texG(src_g,x,y);
    pixR = clampc(round(sum));;
    //Info("%d,%d Red val %f",x,y,sum);

    /* Interpolate Blue */
    if (odd(y)) {
      /* Blue/Green rows */
      /* Use left and right pixels */
      /* Get the difference between the Red and Green
       * channel (use only the sampled Green values) */
      sum = (texB(src_g,x-1,y) - texG(src_g,x-1,y)) +
            (texB(src_g,x+1,y) - texG(src_g,x+1,y));
    } else {
      /* Red/Green rows */
      /* Use top and bottom values */
      sum = (texB(src_g,x,y-1) - texG(src_g,x,y-1)) +
            (texB(src_g,x,y+1) - texG(src_g,x,y+1));
    }
    /* Apply low pass filter */
    sum /= 2.0;
    sum += texG(src_g,x,y);
    pixB = clampc(round(sum));
    //Info("%d,%d pixB : %d , sum %0.2f G:%d",x,y,pixB,sum,texG(src_g,x,y));
  }

  uint dest_width = width - 2*P;
  int dx = x - P;
  int dy = y - P;


#ifndef _TEST
  if (g_tmp_result != NULL) {
    // During testing, skip global memory access
    pixel *res = &g_tmp_result[y * width + x];
    //pixel *res = g_tmp_result + res_index;
    res[R] = pixR;
    res[G] = texG(src_g,x,y);
    res[B] = pixB;
  }
#endif

  //cuCvRGBtoLab(pixR, pixG, pixB, &res_pix->x, &res_pix->y, &res_pix->z);
  // inlining to avoid passing point arguments

  float4 lab;
  float b = pixB/255.0, r = pixR/255.0;
  float g = texG(src_g,x,y)/255.0;
  float x_, y_, z;

  x_ = b*labXb_32f + g*labXg_32f + r*labXr_32f;
  y_ = b*labYb_32f + g*labYg_32f + r*labYr_32f;
  z =  b*labZb_32f + g*labZg_32f + r*labZr_32f;

  if( x_ > labT_32f )
    x_ = cvCbrt(x_);
  else
    x_ = x_*labSmallScale_32f + labSmallShift_32f;

  if( z > labT_32f )
    z = cvCbrt(z);
  else
    z = z*labSmallScale_32f + labSmallShift_32f;

  if( y_ > labT_32f )
  {
    y_ = cvCbrt(y_);
    lab.x = y_*labLScale_32f - labLShift_32f; // L
  }
  else
  {
    lab.x = y_*labLScale2_32f; // L
    y_ = y_*labSmallScale_32f + labSmallShift_32f;
  }

  lab.y = 500.f*(x_ - y_); // a
  lab.z = 200.f*(y_ - z); // b

  g_result[dx + (dy*dest_width)] = lab;
}

__global__ void AddIntsCuda(int * a , int * b)
{
    *a += *b;
}

int main()
{
    int a= 5;
    int  b = 10;
    int * d_a;
    int * d_b;

    hipMalloc(&d_a,sizeof(int));
    hipMalloc(&d_b,sizeof(int));

    hipMemcpy(d_a,&a,sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_b,&b,sizeof(int),hipMemcpyHostToDevice);

    AddIntsCuda<<<1,1>>>(d_a,d_b);

    hipMemcpy(&a,d_a,sizeof(int),hipMemcpyDeviceToHost);

    g_print("Result is %d\n",a);

}
